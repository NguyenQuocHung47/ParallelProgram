
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <random>
#include <algorithm>
#include <cassert>
#include <fstream>
#include <sstream>
#include <stdexcept>
#include <iomanip>
#include <cstring> // For memset and memcpy
#include <stdio.h>
using namespace std;

#define CHECK(call) \
{ \
	const hipError_t error = call; \
	if (error != hipSuccess) \
	{ \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
		fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(EXIT_FAILURE); \
	} \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Function to parse a CSV file into a pointer array
float* load_csv(const string& filepath, int& rows, int& cols, bool normalize = false) {
    ifstream file(filepath);
    if (!file.is_open()) throw runtime_error("Cannot open file: " + filepath);

    vector<vector<float>> data;
    string line;

    while (getline(file, line)) {
        vector<float> row;
        stringstream ss(line);
        string value;

        while (getline(ss, value, ',')) {
            float val = stof(value);
            if (normalize) val /= 255.0; // Normalize pixel values to [0, 1]
            row.push_back(val);
        }
        data.push_back(row);
    }

    rows = data.size();
    cols = data[0].size();
    float* result = new float[rows * cols];
    for (int i = 0; i < rows; ++i) {
        memcpy(result + i * cols, data[i].data(), cols * sizeof(float));
    }

    file.close();
    return result;
}

// Function to load labels and convert to one-hot encoding
float* load_labels_csv(const string& filepath, int& rows, int num_classes = 10) {
    ifstream file(filepath);
    if (!file.is_open()) throw runtime_error("Cannot open file: " + filepath);

    vector<vector<float>> labels;
    string line;

    while (getline(file, line)) {
        int label = stoi(line); // Read the label
        vector<float> one_hot(num_classes, 0.0);
        one_hot[label] = 1.0; // One-hot encode
        labels.push_back(one_hot);
    }

    rows = labels.size();
    float* result = new float[rows * num_classes];
    for (int i = 0; i < rows; ++i) {
        memcpy(result + i * num_classes, labels[i].data(), num_classes * sizeof(float));
    }

    file.close();
    return result;
}

// Main function to load and preprocess the dataset
void load_and_preprocess_dataset_csv(const string& train_images_path,
    const string& train_labels_path,
    const string& test_images_path,
    const string& test_labels_path,
    float*& train_images,
    float*& train_labels,
    float*& test_images,
    float*& test_labels,
    int& train_image_rows, int& train_label_rows,
    int& test_image_rows, int& test_label_rows,
    int& image_cols, int& num_classes) {
    try {
        // Load train images and normalize
        train_images = load_csv(train_images_path, train_image_rows, image_cols, true);

        // Load train labels and one-hot encode
        train_labels = load_labels_csv(train_labels_path, train_label_rows, num_classes);

        // Load test images and normalize
        test_images = load_csv(test_images_path, test_image_rows, image_cols, true);

        // Load test labels and one-hot encode
        test_labels = load_labels_csv(test_labels_path, test_label_rows, num_classes);

        // Verify consistency of data
        if (train_image_rows != train_label_rows) {
            throw runtime_error("Mismatch between number of train images and labels.");
        }
        if (test_image_rows != test_label_rows) {
            throw runtime_error("Mismatch between number of test images and labels.");
        }

        cout << "Dataset loaded successfully:" << endl;
        cout << " - Train samples: " << train_image_rows << endl;
        cout << " - Test samples: " << test_image_rows << endl;

    }
    catch (const exception& ex) {
        cerr << "Error while loading dataset: " << ex.what() << endl;
    }
}

// Activation functions
inline float relu(float x) { return max(0.0f, x); }
inline float relu_derivative(float x) { return x > 0 ? 1.0f : 0.0f; }

void softmax(float* outputs, int size) {
    float max_val = *max_element(outputs, outputs + size);
    float sum = 0.0;
    for (int i = 0; i < size; ++i) {
        outputs[i] = exp(outputs[i] - max_val); // Subtract max_val for numerical stability
        sum += outputs[i];
    }
    for (int i = 0; i < size; ++i) outputs[i] /= sum;
}

// Cross-entropy loss
float cross_entropy_loss(const float* predictions, const float* labels, int size) {
    float loss = 0.0;
    for (int i = 0; i < size; ++i) {
        loss -= labels[i] * log(predictions[i] + 1e-15); // Avoid log(0)
    }
    return loss;
}

// CUDA kernel for forward pass
__constant__ float c_biases[128]; // Adjust size as needed
__constant__ float c_bias_gradients[128]; // Adjust size as needed

__global__ void forward_kernel1(const float* inputs, float* outputs, const float* weights, int input_size, int output_size, int batch_size) {
    extern __shared__ float shared_inputs[];

    int sample_idx = blockIdx.x;
    int neuron_idx = threadIdx.x;
    if (threadIdx.x < input_size) {
        shared_inputs[threadIdx.x] = inputs[sample_idx * input_size + threadIdx.x];
    }
    __syncthreads();
    if (sample_idx < batch_size && neuron_idx < output_size) {
        float sum = c_biases[neuron_idx];
        for (int j = 0; j < input_size; ++j) {
            sum += shared_inputs[j] * weights[neuron_idx * input_size + j];
        }
        outputs[sample_idx * output_size + neuron_idx] = sum;
    }
}


// CUDA kernel for backward pass
__global__ void backward_kernel2(const float* output_gradients, const float* weights,
                                 float* input_gradients, float* weight_gradients,
                                 const float* inputs, float* bias_gradients, int input_size, int output_size, int batch_size) {
    extern __shared__ float shared_output_gradients[];

    int sample = blockIdx.y;
    int input_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadIdx.x < output_size) {
        shared_output_gradients[threadIdx.x] = output_gradients[sample * output_size + threadIdx.x];
    }
    __syncthreads();

    if (input_idx < input_size && sample < batch_size) {
        float gradient = 0.0, input = inputs[sample * input_size + input_idx];
        for (int i = 0; i < output_size; ++i) {
            // Accumulate weight gradients
            atomicAdd(&weight_gradients[i * input_size + input_idx], shared_output_gradients[i] * input);
        }
        if (input > 0) {
            for (int i = 0; i < output_size; ++i) {
                gradient += shared_output_gradients[i] * weights[i * input_size + input_idx];
            }
        }

        // Accumulate input gradients
        input_gradients[sample * input_size + input_idx]=gradient;
    }
    if (threadIdx.x == 0) {
        for (int i = 0; i < output_size; ++i) {
            atomicAdd(&bias_gradients[i], shared_output_gradients[i]);
        }
    }
}

// CUDA kernel for weight updates
__global__ void update_weights_kernel(float* weights, float* biases, const float* weight_gradients,
                                       float lrDivBatchSize, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int neuron = idx / input_size;
    int input = idx % input_size;
    if (input < input_size)
    {
        weights[neuron * input_size + input] -= lrDivBatchSize * weight_gradients[neuron * input_size + input];
    }

    if (idx % input_size == 0)
    {
        biases[neuron] -= lrDivBatchSize * c_bias_gradients[neuron];
    }
}
int num_streams = 4;
hipStream_t* streams = new hipStream_t[num_streams];


class Layer {
public:
    float* weights;
    float* biases;
    float* outputs = nullptr;
    float* inputs = nullptr;
    float* d_input;
    float* d_output;
    float* d_weights;
    float* d_weight_gradients;
    float* d_bias_gradients;
    float* d_batch_input_gradients;
    float* d_biases;
    int input_size, output_size;

    Layer(int input_size, int output_size)
        : input_size(input_size), output_size(output_size) {

        weights = new float[input_size * output_size];
        biases = new float[output_size]();
        //initialize_weights();
        load_weights("init_weight.bin");
        // Allocate streams
    }

    ~Layer() {
        //delete[] inputs;
        hipHostFree(inputs);
        //delete[] outputs;
        hipHostFree(outputs);
        delete[] weights;
        delete[] biases;


        CHECK(hipFree(d_input));
        CHECK(hipFree(d_output));
        CHECK(hipFree(d_weights));
        CHECK(hipFree(d_biases));
        CHECK(hipFree(d_weight_gradients));
        CHECK(hipFree(d_bias_gradients));
        CHECK(hipFree(d_batch_input_gradients));
    }

    void initialize_weights() {
        random_device rd;
        mt19937 gen(rd());
        normal_distribution<> dist(0.0f, 0.1f);
        for (int i = 0; i < input_size * output_size; ++i) {
            weights[i] = dist(gen);
        }
    }

    void cuda_malloc(int batch_size) {
        CHECK(hipMalloc(&d_biases, output_size * sizeof(float)));
        CHECK(hipMalloc(&d_input, batch_size * input_size * sizeof(float)));
        CHECK(hipMalloc(&d_output, batch_size * output_size * sizeof(float)));
        CHECK(hipMalloc(&d_weights, input_size * output_size * sizeof(float)));
        CHECK(hipMalloc(&d_weight_gradients, input_size * output_size * sizeof(float)));
        CHECK(hipMalloc(&d_bias_gradients, output_size * sizeof(float)));
        CHECK(hipMalloc(&d_batch_input_gradients, batch_size * input_size * sizeof(float)));
        CHECK(hipMemcpy(d_weights, weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_biases, biases, output_size * sizeof(float), hipMemcpyHostToDevice));
    }

    void forward_with_streams(const float* batch_input, int batch_size, int num_streams) {
        if (inputs == nullptr) hipHostMalloc(&inputs, batch_size * input_size * sizeof(float), hipHostMallocDefault);
        if (outputs == nullptr) hipHostMalloc(&outputs, batch_size * output_size * sizeof(float), hipHostMallocDefault);
        //CHECK(cudaMalloc(&d_input, batch_size * input_size * sizeof(float)));
        //CHECK(cudaMalloc(&d_output, batch_size * output_size * sizeof(float)));

        memcpy(inputs, batch_input, batch_size * input_size * sizeof(float)); // Copy input data
        CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_biases), d_biases, output_size * sizeof(float)));

        int threads = input_size;
        int blocks = (batch_size + num_streams - 1) / num_streams;
        int shared_memory_size = input_size * sizeof(float);
        int chunkSize = (batch_size + num_streams - 1) / num_streams;

        for (int i = 0; i < num_streams; ++i) {
            int offset = i * chunkSize * input_size;
            int size = min(chunkSize, batch_size - i * chunkSize) * input_size;

            CHECK(hipMemcpyAsync(d_input + offset, inputs + offset, size * sizeof(float), hipMemcpyHostToDevice, streams[i]));
            forward_kernel1<<<blocks, threads, shared_memory_size, streams[i]>>>(d_input + offset, d_output + i*chunkSize*output_size, d_weights, input_size, output_size, chunkSize);
            CHECK(hipMemcpyAsync(outputs + i*chunkSize*output_size, d_output + i*chunkSize*output_size, chunkSize*output_size * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
        }

        for (int i = 0; i < num_streams; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }

        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        //CHECK(cudaFree(d_input));
        //CHECK(cudaFree(d_output));
    }

    void forward(const float* batch_input, int batch_size) {
        const int num_streams = 4; // Fixed number of streams
        if (num_streams > 1) {
            forward_with_streams(batch_input, batch_size, num_streams);
        } else {
            if (inputs == nullptr) inputs = new float[batch_size * input_size];

            CHECK(hipMalloc(&d_input, batch_size * input_size * sizeof(float)));
            CHECK(hipMalloc(&d_output, batch_size * output_size * sizeof(float)));

            memcpy(inputs, batch_input, batch_size * input_size * sizeof(float)); // Copy input data
            CHECK(hipMemcpy(d_input, inputs, batch_size * input_size * sizeof(float), hipMemcpyHostToDevice));
            CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_biases), d_biases, output_size * sizeof(float)));

            int threads = input_size;
            int blocks = batch_size;
            int shared_memory_size = input_size * sizeof(float);

            forward_kernel1<<<blocks, threads, shared_memory_size>>>(d_input, d_output, d_weights, input_size, output_size, batch_size);

            CHECK(hipDeviceSynchronize());

            if (outputs == nullptr) outputs = new float[batch_size * output_size];
            CHECK(hipMemcpy(outputs, d_output, batch_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
            CHECK(hipFree(d_input));
            CHECK(hipFree(d_output));
        }
    }

    void backward_with_streams(const float* batch_output_gradients, int batch_size, int num_streams) {
        //CHECK(cudaMalloc(&d_input, batch_size * input_size * sizeof(float)));
        //CHECK(cudaMalloc(&d_output, batch_size * output_size * sizeof(float)));
        //CHECK(cudaMemcpy(d_output, batch_output_gradients, batch_size * output_size * sizeof(float), cudaMemcpyHostToDevice));
        //CHECK(cudaMemcpy(d_input, inputs, batch_size * input_size * sizeof(float), cudaMemcpyHostToDevice));
        hipMemset(d_weight_gradients, 0, input_size * output_size * sizeof(float));
        hipMemset(d_bias_gradients, 0, output_size * sizeof(float));

        int chunkSize = (batch_size + num_streams - 1) / num_streams;
        dim3 threads(input_size);
        dim3 blocks(1, chunkSize);

        for (int i = 0; i < num_streams; ++i) {
            int offset = i * chunkSize * input_size;
            int size = min(chunkSize, batch_size - i * chunkSize) * input_size;
            CHECK(hipMemcpyAsync(d_output + i * chunkSize * output_size, batch_output_gradients + i * chunkSize * output_size, chunkSize* output_size * sizeof(float), hipMemcpyHostToDevice, streams[i]));
            //CHECK(cudaMemcpyAsync(d_input + offset, inputs + offset, size * sizeof(float), cudaMemcpyHostToDevice, streams[i]));
            backward_kernel2<<<blocks, threads, output_size * sizeof(float), streams[i]>>>(
                d_output + i * chunkSize * output_size, d_weights, d_input + offset, d_weight_gradients,
                d_input + offset, d_bias_gradients, input_size, output_size, chunkSize);
            CHECK(hipMemcpyAsync(inputs + offset, d_input + offset, size * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
        }

        for (int i = 0; i < num_streams; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }

        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(inputs, d_input, batch_size * input_size * sizeof(float), hipMemcpyDeviceToHost));
        //CHECK(cudaFree(d_input));
        //CHECK(cudaFree(d_output));
    }

    void backward(const float* batch_output_gradients, int batch_size) {
        const int num_streams = 4; // Fixed number of streams
        if (num_streams > 1) {
            backward_with_streams(batch_output_gradients, batch_size, num_streams);
        } else {
            CHECK(hipMalloc(&d_input, batch_size * input_size * sizeof(float)));
            CHECK(hipMalloc(&d_output, batch_size * output_size * sizeof(float)));
            CHECK(hipMemcpy(d_output, batch_output_gradients, batch_size * output_size * sizeof(float), hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_input, inputs, batch_size * input_size * sizeof(float), hipMemcpyHostToDevice));
            hipMemset(d_weight_gradients, 0, input_size * output_size * sizeof(float));
            hipMemset(d_bias_gradients, 0, output_size * sizeof(float));

            dim3 blocks(1, batch_size);
            dim3 threads(input_size);

            backward_kernel2<<<blocks, threads, output_size * sizeof(float)>>>(d_output, d_weights, d_input, d_weight_gradients, d_input, d_bias_gradients, input_size, output_size, batch_size);

            CHECK(hipGetLastError());
            CHECK(hipDeviceSynchronize());

            CHECK(hipMemcpy(inputs, d_input, batch_size * input_size * sizeof(float), hipMemcpyDeviceToHost));
            CHECK(hipFree(d_input));
            CHECK(hipFree(d_output));
        }
    }

    void update_weights(float learning_rate, int batch_size) {
        float lrDivBatchSize = learning_rate / batch_size;
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_bias_gradients), d_bias_gradients, output_size * sizeof(float)));

        dim3 threads(128);
        dim3 blocks((output_size * input_size + threads.x - 1) / threads.x);
        update_weights_kernel<<<blocks, threads>>>(d_weights, d_biases, d_weight_gradients, lrDivBatchSize, input_size, output_size);

        CHECK(hipDeviceSynchronize());
    }

    void save_bias() {
        CHECK(hipMemcpy(biases, d_biases, output_size * sizeof(float), hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(weights, d_weights, input_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
    }

    void load_weights(const string& filepath) {
        ifstream file(filepath, ios::binary);
        if (!file.is_open()) throw runtime_error("Cannot open file to load weights: " + filepath);

        // Load weights
        file.read(reinterpret_cast<char*>(weights), input_size * output_size * sizeof(float));

        // Load biases
        file.read(reinterpret_cast<char*>(biases), output_size * sizeof(float));

        file.close();
        cout << "Weights loaded from " << filepath << endl;
    }
};

class NeuralNetwork {
public:
    Layer** layers;
    int num_layers;

    NeuralNetwork(const vector<int>& architecture) {
        num_layers = architecture.size() - 1;
        layers = new Layer * [num_layers];
        for (int i = 0; i < num_layers; ++i) {
            layers[i] = new Layer(architecture[i], architecture[i + 1]);
        }
    }

    ~NeuralNetwork() {
        for (int i = 0; i < num_layers; ++i) {
            delete layers[i];
        }
        delete[] layers;
    }

    void cuda_malloc(int batch_size) {
        for (int i = 0; i < num_layers; ++i) {
            layers[i]->cuda_malloc(batch_size);
        }
    }

    float* forward(const float* batch_input, int batch_size) {
        float* activations = const_cast<float*>(batch_input);
        for (int i = 0; i < num_layers; ++i) {
            layers[i]->forward(activations, batch_size);
            activations = layers[i]->outputs;
            if (i < num_layers - 1) {
                for (int j = 0; j < batch_size * layers[i]->output_size; ++j) {
                    activations[j] = relu(activations[j]);
                }
            }
            else {
                for (int j = 0; j < batch_size; ++j) {
                    softmax(activations + j * layers[i]->output_size, layers[i]->output_size);
                }
            }
        }
        return activations;
    }

    void backward(const float* predictions, const float* labels, int batch_size, int num_classes) {
        float* batch_output_gradients = new float[batch_size * num_classes];
        for (int sample = 0; sample < batch_size; ++sample) {
            for (int i = 0; i < num_classes; ++i) {
                batch_output_gradients[sample * num_classes + i] = predictions[sample * num_classes + i] - labels[sample * num_classes + i];
            }
        }

        for (int i = num_layers - 1; i >= 0; --i) {
            layers[i]->backward(batch_output_gradients, batch_size);
            batch_output_gradients = layers[i]->inputs;
        }
    }

    void update_weights(float learning_rate, int batch_size) {
        for (int i = 0; i < num_layers; ++i) {
            layers[i]->update_weights(learning_rate, batch_size);
        }
    }

    void update_bias() {
        for (int i = 0; i < num_layers; ++i) {
            layers[i]->save_bias();
        }
    }

    void train(const float* data, const float* labels, int num_samples, int batch_size, int num_classes, int epochs, float learning_rate) {
        cuda_malloc(batch_size);
        for (int epoch = 1; epoch <= epochs; ++epoch) {
            float total_loss = 0.0;
            for (int i = 0; i < num_samples; i += batch_size) {
                int current_batch_size = min(batch_size, num_samples - i);
                const float* batch_data = data + i * layers[0]->input_size;
                const float* batch_labels = labels + i * num_classes;

                float* predictions = forward(batch_data, current_batch_size);
                for (int j = 0; j < current_batch_size; ++j) {
                    total_loss += cross_entropy_loss(predictions + j * num_classes, batch_labels + j * num_classes, num_classes);
                }
                backward(predictions, batch_labels, current_batch_size, num_classes);
                update_weights(learning_rate, current_batch_size);
            }
            cout << "Epoch " << epoch << " - Loss: " << total_loss / num_samples << endl;
        }
        update_bias();
    }

    // Save the model to a file
    void save_model(const string& filepath) {
        ofstream file(filepath, ios::binary);
        if (!file.is_open()) throw runtime_error("Cannot open file to save model: " + filepath);
        // Save the number of layers
        file.write(reinterpret_cast<char*>(&num_layers), sizeof(num_layers));

        for (int i = 0; i < num_layers; ++i) {
            Layer* layer = layers[i];

            // Save input_size and output_size
            file.write(reinterpret_cast<char*>(&layer->input_size), sizeof(layer->input_size));
            file.write(reinterpret_cast<char*>(&layer->output_size), sizeof(layer->output_size));

            // Save weights
            file.write(reinterpret_cast<char*>(layer->weights), layer->input_size * layer->output_size * sizeof(float));

            // Save biases
            file.write(reinterpret_cast<char*>(layer->biases), layer->output_size * sizeof(float));
        }

        file.close();
        cout << "Model saved to " << filepath << endl;
    }

    // Load the model from a file
    void load_model(const string& filepath) {
        ifstream file(filepath, ios::binary);
        if (!file.is_open()) throw runtime_error("Cannot open file to load model: " + filepath);

        // Read the number of layers
        int saved_num_layers;
        file.read(reinterpret_cast<char*>(&saved_num_layers), sizeof(saved_num_layers));

        // Ensure the architecture matches the saved model
        if (saved_num_layers != num_layers) {
            throw runtime_error("Saved model architecture does not match the current network.");
        }

        for (int i = 0; i < num_layers; ++i) {
            Layer* layer = layers[i];

            // Read input_size and output_size
            int saved_input_size, saved_output_size;
            file.read(reinterpret_cast<char*>(&saved_input_size), sizeof(saved_input_size));
            file.read(reinterpret_cast<char*>(&saved_output_size), sizeof(saved_output_size));

            if (saved_input_size != layer->input_size || saved_output_size != layer->output_size) {
                throw runtime_error("Layer dimensions do not match the saved model.");
            }

            // Load weights
            file.read(reinterpret_cast<char*>(layer->weights), layer->input_size * layer->output_size * sizeof(float));

            // Load biases
            file.read(reinterpret_cast<char*>(layer->biases), layer->output_size * sizeof(float));
        }

        file.close();
        cout << "Model loaded from " << filepath << endl;
    }
};

int main(int argc, char* argv[]) {
    int batch_size = 32;
    if (argc > 1) {
        batch_size = atoi(argv[1]);
    }

    float* train_images;
    float* train_labels;
    float* test_images;
    float* test_labels;

    int train_image_rows, train_label_rows;
    int test_image_rows, test_label_rows;
    int image_cols, num_classes = 10;

    const string train_images_path = "x_train.csv";
    const string train_labels_path = "y_train.csv";
    const string test_images_path = "x_test.csv";
    const string test_labels_path = "y_test.csv";

    // Load and preprocess the dataset
    load_and_preprocess_dataset_csv(train_images_path, train_labels_path, test_images_path, test_labels_path,
        train_images, train_labels, test_images, test_labels,
        train_image_rows, train_label_rows, test_image_rows, test_label_rows,
        image_cols, num_classes);

    // Define network architecture and create neural network
    vector<int> architecture = { image_cols, 128, 128, num_classes };
    NeuralNetwork nn(architecture);
    for (int i = 0; i < num_streams; ++i) {
        CHECK(hipStreamCreate(&streams[i]));
    }
    GpuTimer timer;
    timer.Start();
    nn.train(train_images, train_labels, train_image_rows, batch_size, num_classes, 10, 0.01f);
    timer.Stop();
    float time = timer.Elapsed();
    cout << "Processing time: " << time << " ms" << endl;

    nn.save_model("model.bin");
    NeuralNetwork nn1(architecture);

    nn1.load_model("model.bin");



    // Evaluate accuracy
    nn1.cuda_malloc(test_image_rows);
    float* test_predictions = nn1.forward(test_images, test_image_rows);
    int correct = 0;
    for (int i = 0; i < test_image_rows; ++i) {
        int predicted = max_element(test_predictions + i * num_classes, test_predictions + (i + 1) * num_classes) - (test_predictions + i * num_classes);
        int actual = max_element(test_labels + i * num_classes, test_labels + (i + 1) * num_classes) - (test_labels + i * num_classes);
        if (predicted == actual) ++correct;
    }
    cout << "Test Accuracy: " << (correct / static_cast<float>(test_image_rows)) * 100.0f << "%" << endl;

    // Clean up dynamically allocated memory
    delete[] train_images;
    delete[] train_labels;
    delete[] test_images;
    delete[] test_labels;
    for (int i = 0; i < num_streams; ++i) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    delete[] streams;

    return 0;
}